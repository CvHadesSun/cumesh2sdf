#include "hip/hip_runtime.h"
#include "main.cuh"
#include <fstream>
#include <iterator>
#include <vector>
#include <chrono>
#include <iostream>


int main(int argc, char ** argv)
{
    std::chrono::high_resolution_clock clock;
    auto start = clock.now();
    std::ifstream fi(argv[argc - 1]);
    if (!fi)
    {
        std::cerr << "[Error] Cannot open input file" << std::endl;
        return 1;
    }
    int F;
    fi >> F;
    // copies all data into buffer
    auto trisCPU = std::make_unique<float3[]>(F * 3);
    float3 * tris;
    hipMalloc(&tris, F * 3 * sizeof(float3));

    for (int i = 0; i < F; i++)
        for (int j = 0; j < 3; j++)
            fi >> trisCPU[3 * i + j].x >> trisCPU[3 * i + j].y >> trisCPU[3 * i + j].z;
    fi.close();
    CHECK_CUDA(hipMemcpy(tris, trisCPU.get(), sizeof(float3) * F * 3, hipMemcpyHostToDevice));
    const auto inputPhase = clock.now() - start;
    start = clock.now();

    // for (int i = 0; i < buffer.size() / 3 / sizeof(float); i++)
    //     printf("%.2f %.2f %.2f\n", tris[i].x, tris[i].y, tris[i].z);

    constexpr const int N = 128;
    RasterizeResult rast = rasterize_tris(tris, F, N, 3.0f / N, 65536, false);
    const auto rasterizePhase = clock.now() - start;
    start = clock.now();
    fill_signs(tris, N, rast, false);
    const auto signPhase = clock.now() - start;
    start = clock.now();
    auto gridDistCPU = std::make_unique<float[]>(N * N * N);
    CHECK_CUDA(hipMemcpy(gridDistCPU.get(), rast.gridDist, sizeof(float) * N * N * N, hipMemcpyDeviceToHost));
    static char _output_buf[2 * 1024 * 1024];
    std::ofstream fo("output.txt", std::fstream::trunc);
    fo.rdbuf()->pubsetbuf(_output_buf, 2 * 1024 * 1024);
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < N; k++)
                fo << gridDistCPU[i * N * N + j * N + k] << '\t';
            fo << '\n';
        }
    fo.close();
    hipFree(tris);
    rast.free();
    const auto outputPhase = clock.now() - start;
    start = clock.now();

    std::clog << "[Timing] Input phase: " << (int)(inputPhase.count() / 1e6) << " ms" << std::endl;
    std::clog << "         Rasterize phase: " << (int)(rasterizePhase.count() / 1e6) << " ms" << std::endl;
    std::clog << "         Sign phase: " << (int)(signPhase.count() / 1e6) << " ms" << std::endl;
    std::clog << "         Output phase: " << (int)(outputPhase.count() / 1e6) << " ms" << std::endl;
    return 0;
}
